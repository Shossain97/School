#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include "../ImageWriter/ImageWriter.h"
#include <fstream>
#include <string>
#include <iostream>
#include <stdlib.h>

int aNum;
// __global__
// __device__
// __host__
//takes from project guidelines page
__host__ void writeTheFile(std::string fName, int xres, int yres, const unsigned char* imageBytes)
{
	unsigned char* row = new unsigned char[3*xres];
	ImageWriter* w = ImageWriter::create(fName,xres,yres);
	int next = 0;
	for (int r=0 ; r<yres ; r++)
	{
		for (int c=0 ; c<3*xres ; c+=3)
		{
			row[c] = row[c+1] = row[c+2] = imageBytes[next++];
		}
		w->addScanLine(row);
	}
	w->closeImageFile();
	delete w;
	delete [] row;
}


__device__ int getLocation3d(int cur_row, int cur_col, int cur_sheet, int rows, int cols, int sheets){

	return (cur_sheet*cols*rows) + (rows*cur_col) +cur_row;

}
__device__ int getLocation2d(int cur_row, int cur_col, int rows, int cols){
	//return((cur_col*rows) +cur_row);
  return ((cur_row*cols)+cur_col);
}
__global__ void calculateMaxAndWorkingBuffer1(int rows, int cols, int sheets, int projectionType, unsigned char* voxelData, unsigned char* maxBuffer, float* workingBuffer){
		int curRow;
		int curCol;
		int curSheet;
		//threads per block = cols
		//blocks per grid = rows

		curCol = blockIdx.x;
		curRow = threadIdx.x;

		int loc2d = getLocation2d(curRow, curCol, rows, cols);
		//int loc3d = getLocation3d(curRow,curCol, i, rows, cols, sheets);

      maxBuffer[loc2d]=0;
      workingBuffer[loc2d]=0;
			unsigned char max=0;
      for(int i=0;i<sheets;i++){
        if(voxelData[getLocation3d(curRow, curCol, i, rows, cols, sheets)]>max){
          max = voxelData[getLocation3d(curRow,curCol, i, rows, cols, sheets)];
        }
        workingBuffer[loc2d]+=((1.0+i)/sheets)*(voxelData[getLocation3d(curRow, curCol, i, rows, cols, sheets)]);
      }
			maxBuffer[loc2d]=max;
		}
__global__	void calculateMaxAndWorkingBuffer2(int rows, int cols, int sheets, int projectionType, unsigned char* voxelData, unsigned char* maxBuffer, float* workingBuffer){
	int curRow;
	int curCol;
	int curSheet;
	// int threadX= blockIdx.y *blockDim.y + threadIdx.y;
	// int threadY = blockIdx.x * blockDim.x + threadIdx.x;
	curCol = blockIdx.x;
	curRow = threadIdx.x;
	int loc2d = getLocation2d(curRow, cols-curCol, rows, cols);
	maxBuffer[loc2d]=0;
	workingBuffer[loc2d]=0;
	unsigned char max=0;
	for(int i=sheets-1;i>=0;i--){
		if(voxelData[getLocation3d(curRow, curCol, i, rows, cols, sheets)]>max){
			max = voxelData[getLocation3d(curRow,curCol, i, rows, cols, sheets)];
		}
		workingBuffer[loc2d]+=((1.0+i)/sheets)*(voxelData[getLocation3d(curRow, curCol, i, rows, cols, sheets)]);
	}
	maxBuffer[loc2d]=max;
}
__global__	void calculateMaxAndWorkingBuffer3(int rows, int cols, int sheets, int projectionType, unsigned char* voxelData, unsigned char* maxBuffer, float* workingBuffer){
	int curRow=0;
	int curCol=0;
	int curSheet=0;
	//threads per block = cols
	//blocks per grid = rows
	// curCol = blockIdx.x;
	// curRow = threadIdx.x;
	curSheet = blockIdx.x;
	curRow = threadIdx.x;

	int loc2d = getLocation2d(curRow, curSheet, rows, sheets);
	//int loc3d = getLocation3d(curRow,curCol, i, rows, cols, sheets);

		maxBuffer[loc2d]=0;
		workingBuffer[loc2d]=0;
		unsigned char max=0;
		for(curCol=cols-1;curCol>=0;curCol--){
			if(voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)]>max){
				max = voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)];
			}
			workingBuffer[loc2d]+=((1.0+curCol)/cols)*(voxelData[getLocation3d(curRow, curCol, curSheet, rows, cols, sheets)]);
		}
		maxBuffer[loc2d]=max;
}
__global__	void calculateMaxAndWorkingBuffer4(int rows, int cols, int sheets, int projectionType, unsigned char* voxelData, unsigned char* maxBuffer, float* workingBuffer){
	int curRow=0;
	int curCol=0;
	int curSheet=0;
	//threads per block = cols
	//blocks per grid = rows
	// curCol = blockIdx.x;
	// curRow = threadIdx.x;
	curSheet = blockIdx.x;
	curRow = threadIdx.x;

	int loc2d = getLocation2d(curRow, sheets-curSheet, rows, sheets);
	//int loc3d = getLocation3d(curRow,curCol, i, rows, cols, sheets);

		maxBuffer[loc2d]=0;
		workingBuffer[loc2d]=0;
		unsigned char max=0;
		for(curCol=0;curCol<cols;curCol++){
			if(voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)]>max){
				max = voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)];
			}
			workingBuffer[loc2d]+=((1.0+curCol)/cols)*(voxelData[getLocation3d(curRow, curCol, curSheet, rows, cols, sheets)]);
		}
		maxBuffer[loc2d]=max;
}
__global__	void calculateMaxAndWorkingBuffer5(int rows, int cols, int sheets, int projectionType, unsigned char* voxelData, unsigned char* maxBuffer, float* workingBuffer){
	int curRow=0;
	int curCol=0;
	int curSheet=0;
	//threads per block = cols
	//blocks per grid = rows
	// curCol = blockIdx.x;
	// curRow = threadIdx.x;
	curCol = blockIdx.x;
	curSheet = threadIdx.x;

	int loc2d = getLocation2d(curSheet, curCol, sheets, cols);
	//int loc3d = getLocation3d(curRow,curCol, i, rows, cols, sheets);

		maxBuffer[loc2d]=0;
		workingBuffer[loc2d]=0;
		unsigned char max=0;
		for(curRow=rows-1;curRow>=0;curRow--){
			if(voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)]>max){
				max = voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)];
			}
			workingBuffer[loc2d]+=((1.0+curRow)/rows)*(voxelData[getLocation3d(curRow, curCol, curSheet, rows, cols, sheets)]);
		}
		maxBuffer[loc2d]=max;
}
__global__	void calculateMaxAndWorkingBuffer6(int rows, int cols, int sheets, int projectionType, unsigned char* voxelData, unsigned char* maxBuffer, float* workingBuffer){
	int curRow=0;
	int curCol=0;
	int curSheet=0;
	//threads per block = cols
	//blocks per grid = rows
	// curCol = blockIdx.x;
	// curRow = threadIdx.x;
	curCol = blockIdx.x;
	curSheet = threadIdx.x;

	int loc2d = getLocation2d(sheets-curSheet, curCol, sheets, cols);
	//int loc3d = getLocation3d(curRow,curCol, i, rows, cols, sheets);

		maxBuffer[loc2d]=0;
		workingBuffer[loc2d]=0;
		unsigned char max=0;
		for(curRow=0;curRow<rows;curRow++){
			if(voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)]>max){
				max = voxelData[getLocation3d(curRow,curCol, curSheet, rows, cols, sheets)];
			}
			workingBuffer[loc2d]+=((1.0+curRow)/rows)*(voxelData[getLocation3d(curRow, curCol, curSheet, rows, cols, sheets)]);
		}
		maxBuffer[loc2d]=max;
}

__global__ void calculateSum(int dimX, int dimY,  float* workingBuffer, unsigned char* sumBuffer, float* maxWeightedSum){
	int rows=dimX;
	int cols = dimY;
	int curRow = threadIdx.x;
	int curCol = blockIdx.x;
	sumBuffer[getLocation2d(curRow, curCol, rows, cols)]=round((workingBuffer[getLocation2d(curRow, curCol, rows, cols)]/maxWeightedSum[0])*255.0);
}
__global__ void getMax(float* max, float* workingBuffer, int totalBuffer){
	max[0]=0;
		for(int i =0;i<totalBuffer;i++){
			if(workingBuffer[i]>max[0]){
				max[0]=workingBuffer[i];
			}
		}
}
int main(int argc, char** argv){
  if(argc!=7){
    printf("You input %d arguments you must input in this format:\n./project3 nRows nCols nSheets fileName projectionType outputFileNameBase\n", argc);
    return 0;
  }
  //make sure CUDA wass compiled properly
  int driverVersion, runtimeVersion;
	hipError_t dv = hipDriverGetVersion(&driverVersion);
	hipError_t rv = hipRuntimeGetVersion(&runtimeVersion);
	// std::cout<< "Driver version: " << driverVersion << "; Runtime version: "
	//      << runtimeVersion << "\n\n";
  //all the dimension data
  int rows, cols, sheets;
  rows=atoi(argv[1]);
  cols=atoi(argv[2]);
  sheets=atoi(argv[3]);
  int amountOfData=rows*cols*sheets;

  //stream file binary into a buffer
  unsigned char* voxelData = new unsigned char[amountOfData];
  std::ifstream istream(argv[4]);
  istream.read(reinterpret_cast<char*>(voxelData),amountOfData);
  //figure out the orientation method
  int projectionType = atoi(argv[5]);
  //depending on the projection type we will allocate a x *y spaced array
  // on to the GPU
  int dimensionsMult;
  //we want at least as many threads as there are pixels for the 2d side we are
  //not traversing
  //
  //face will be x*y dimension
  //so
  int blocksPerGrid;
  int threadsPerBlock;

  if(projectionType==1||projectionType==2){
    dimensionsMult=rows* cols;
    blocksPerGrid=cols;
    threadsPerBlock = rows;
  }
  else if(projectionType==3||projectionType==4){
    dimensionsMult = rows * sheets;
    blocksPerGrid = sheets;
    threadsPerBlock = rows;
  }
  else if(projectionType==5||projectionType==6){
    dimensionsMult = cols * sheets;
    blocksPerGrid = cols;
    threadsPerBlock = sheets;
  }
  else{
    printf("broke\n");
    return 0;
  }
	dim3 tPB(threadsPerBlock);
	dim3 bPG(blocksPerGrid);
  size_t maxBufferSize = dimensionsMult * (sizeof(unsigned char));
  size_t workingBufferSize= dimensionsMult* (sizeof(float));
  size_t dataSize = amountOfData * (sizeof(unsigned char));
  unsigned char* host_maxBuffer= new unsigned char[dimensionsMult];
	unsigned char* host_sumBuffer= new unsigned char[dimensionsMult];
  // int threadsNeeded=dimensionsMult;


 //allocate space on GPU
 unsigned char* maxBuffer;
 unsigned char* dataBuffer;
 float* workingBuffer;
 unsigned char* sumBuffer;
 hipMalloc((void**)&maxBuffer, maxBufferSize);
 hipMalloc((void**)&dataBuffer, dataSize);
 hipMalloc((void**)&workingBuffer, workingBufferSize);
 hipMalloc((void**)&sumBuffer, maxBufferSize);
 float* h_Max = new float[1];
 h_Max[0]=0.0;
 float* max;
 hipMalloc((void**)&max, sizeof(float));

//copy the voxel data over to device
 hipMemcpy(dataBuffer, voxelData, dataSize, hipMemcpyHostToDevice);
 hipMemcpy(max, h_Max, sizeof(float), hipMemcpyHostToDevice);


 //invoke kernel

// hipMalloc((void**)&d_index, sizeof(int));
switch(projectionType){
	case 1:
		calculateMaxAndWorkingBuffer1<<<bPG, tPB>>>(rows, cols, sheets, projectionType, dataBuffer, maxBuffer, workingBuffer);
		break;
	case 2:
		calculateMaxAndWorkingBuffer2<<<blocksPerGrid, threadsPerBlock>>>(rows, cols, sheets, projectionType, dataBuffer, maxBuffer, workingBuffer);
		break;
	case 3:
		calculateMaxAndWorkingBuffer3<<<blocksPerGrid, threadsPerBlock>>>(rows, cols, sheets, projectionType, dataBuffer, maxBuffer, workingBuffer);
		break;
	case 4:
		calculateMaxAndWorkingBuffer4<<<blocksPerGrid, threadsPerBlock>>>(rows, cols, sheets, projectionType, dataBuffer, maxBuffer, workingBuffer);
		break;
	case 5:
		calculateMaxAndWorkingBuffer5<<<blocksPerGrid, threadsPerBlock>>>(rows, cols, sheets, projectionType, dataBuffer, maxBuffer, workingBuffer);
		break;
	case 6:
		calculateMaxAndWorkingBuffer6<<<blocksPerGrid, threadsPerBlock>>>(rows, cols, sheets, projectionType, dataBuffer, maxBuffer, workingBuffer);
		break;

}


 //get stuff back
hipDeviceSynchronize();
hipMemcpy(host_maxBuffer, maxBuffer, maxBufferSize, hipMemcpyDeviceToHost);
hipDeviceSynchronize();
getMax<<<1,1>>>(max, workingBuffer, blocksPerGrid*threadsPerBlock);
// hipDeviceSynchronize();
// hipMemcpy(h_Max, max, sizeof(float), hipMemcpyDeviceToHost);
// calculateSum<<<blocksPerGrid, threadsPerBlock>>>(threadsPerBlock, blocksPerGrid, workingBuffer, sumBuffer);
// std::cout<<"Max: "<<h_Max[0]<<"\n";
hipDeviceSynchronize();
//void calculateSum(int dimX, int dimY,  float* workingBuffer, unsigned char* maxWeightedSum, float* sumBuffer)
calculateSum<<<blocksPerGrid, threadsPerBlock>>>(threadsPerBlock, blocksPerGrid, workingBuffer, sumBuffer, max);
hipDeviceSynchronize();
hipMemcpy(host_sumBuffer, sumBuffer, maxBufferSize, hipMemcpyDeviceToHost);


// hipMemcpy( &h_index , d_index, sizeof(int), hipMemcpyDeviceToHost);


 //run next kernel

 //get stuff back


 //out put images

 std::string param(argv[6]);
 param.append("MAX.png");
 // printf(param.c_str());

 writeTheFile(param,blocksPerGrid, threadsPerBlock, host_maxBuffer);

 std::string sumParam(argv[6]);
 sumParam.append("SUM.png");
 writeTheFile(sumParam, blocksPerGrid, threadsPerBlock, host_sumBuffer);
 hipFree(dataBuffer);
 hipFree(maxBuffer);
 hipFree(workingBuffer);
 delete[] voxelData;
 delete[] host_maxBuffer;
  return 0;
}
